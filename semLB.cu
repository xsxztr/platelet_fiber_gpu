/*
 semLB.cu
 
 Main functions for running the GPU kernels.

 Author: Scott Christley <schristley@mac.com>
 
 Copyright (C) 2010 Scott Christley
 All rights reserved.

 Redistribution and use in source and binary forms, with or without
 modification, are permitted provided that the following conditions
 are met: 1. Redistributions of source code must retain the above
 copyright notice, this list of conditions and the following
 disclaimer.  2. Redistributions in binary form must reproduce the
 above copyright notice, this list of conditions and the following
 disclaimer in the documentation and/or other materials provided with
 the distribution.  3. The name of the author may not be used to
 endorse or promote products derived from this software without
 specific prior written permission.

 THIS SOFTWARE IS PROVIDED BY THE AUTHOR ``AS IS'' AND ANY EXPRESS OR
 IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 ARE DISCLAIMED.  IN NO EVENT SHALL THE AUTHOR BE LIABLE FOR ANY
 DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE
 GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER
 IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
 OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN
 IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

 */

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>
#include <gsl/gsl_rng.h>


// Constant GPU array for holding parameters
__constant__ float model_Parameters[100];

extern "C" {
  #define X_a 64
  #define Y_a 1
  #include "GPUDefines.h"

  #include <stdio.h>
  //#include <BioSwarm/GPUDefines.h>

typedef struct _fluid_GPUgrids {
	float dt;
    	float dx;
    	int width;
    	int height;
    	int depth;
    	size_t pitch;
    	hipExtent iextent; //int extent
    	hipExtent fextent; //float extent
    	hipExtent dextent; //double extent
   	hipPitchedPtr fIN[19];
    	hipPitchedPtr fOUT[19];
    	hipPitchedPtr ux;
    	hipPitchedPtr uy;
    	hipPitchedPtr uz;
    	hipPitchedPtr obst;
    	hipPitchedPtr rho;
    	hipPitchedPtr Fx;  //External force
    	hipPitchedPtr Fy;
    	hipPitchedPtr Fz;
    	hipPitchedPtr vWFbond;
    	void *deviceStruct;
  } fluid_GPUgrids;


  #include "sem_kernel.cu"
  #include "LB_kernel.cu"

 

void cudacheck(const char *message)
{
    	hipError_t error = hipGetLastError();
    	if (error!=hipSuccess)
    	{
      		printf("cudaERROR: %s : %i (%s)\n", message, error, hipGetErrorString(error));
      		exit(EXIT_FAILURE);
    	}
}
  
void pitchcheck(size_t pitch_test, size_t pitch, int id)
{
    	if (pitch_test != pitch)
	{ 
        	printf("pitch is not match at %d.\n", id);
         	exit(1); 
       	}
    
}


////////// GPU for Fluid ////////////////////////////////////////////////////////////////////////
   ///BEGIN: FLUID ALLOC AND INITIAL ///

  // Allocation
void *fluid_allocGPUKernel(void *model, float dt, float dx, int width, int height, int depth)
{
    	fluid_GPUgrids *grids = (fluid_GPUgrids *)malloc(sizeof(fluid_GPUgrids));
    	int i;

    // Save parameters
    	grids->dt = dt;
    	grids->dx = dx;
    	grids->width = width;
    	grids->height = height;
    	grids->depth = depth;

    // Allocate device memory
    	grids->iextent = make_hipExtent(grids->width*sizeof(int), grids->height, 4);
    	grids->fextent = make_hipExtent(grids->width*sizeof(float), grids->height, grids->depth);
    	grids->dextent = make_hipExtent(grids->width*sizeof(double), grids->height, grids->depth);
    	for (i = 0; i < 19; ++i) 
	{
      		hipMalloc3D(&(grids->fIN[i]), grids->dextent);
      		cudacheck("FIN alloc");
      //printf("%d\n", grids->fIN[i].pitch);
      		hipMalloc3D(&(grids->fOUT[i]), grids->dextent);
      		cudacheck("FOUT alloc");
      //printf("%d\n", grids->fIN[i].pitch);
    	}
    
    	hipMalloc3D(&(grids->vWFbond), grids->iextent);
    	hipMalloc3D(&(grids->ux), grids->dextent);
    	hipMalloc3D(&(grids->uy), grids->dextent);
   	hipMalloc3D(&(grids->uz), grids->dextent);
    	hipMalloc3D(&(grids->obst), grids->fextent);
    	hipMalloc3D(&(grids->rho), grids->dextent);
    	hipMalloc3D(&(grids->Fx), grids->dextent);
    	hipMalloc3D(&(grids->Fy), grids->dextent);
    	hipMalloc3D(&(grids->Fz), grids->dextent);

    	hipMalloc(&(grids->deviceStruct), sizeof(fluid_GPUgrids));
    	cudacheck("Fluid alloc");

    	return grids;
}

  // Initialization
void fluid_initGPUKernel(void *model, void *g, int aFlag, float *hostParameters, void **fIN, void **fOUT, void *ux,
			   void *uy, void *uz, void *rho, void *obstacle, void *Fx, void *Fy, void *Fz, void *vWFbond)
{
    	fluid_GPUgrids *grids = (fluid_GPUgrids *)g;
    	int i;

    	if (aFlag) {
      // Copy host memory to device memory
     // hipMemcpyToSymbol(HIP_SYMBOL(model_Parameters), hostParameters, 16 * sizeof(float), 0, hipMemcpyHostToDevice);
     // cudacheck("constant");

      		hipMemcpy3DParms p3d = {0};
      		hipError_t error;
      		p3d.extent = grids->dextent;
      		p3d.kind = hipMemcpyHostToDevice;

      		for (i = 0; i < 19; ++i) {
			p3d.srcPtr = make_hipPitchedPtr(fIN[i], grids->width*sizeof(double), grids->width*sizeof(double), grids->height);
			p3d.dstPtr = grids->fIN[i];
			error = hipMemcpy3D(&p3d);
		if (error) printf("cudaERROR in: %s\n", hipGetErrorString(error));

		p3d.srcPtr = make_hipPitchedPtr(fOUT[i], grids->width*sizeof(double), grids->width*sizeof(double), grids->height);
		p3d.dstPtr = grids->fOUT[i];
		error = hipMemcpy3D(&p3d);
		if (error) printf("cudaERROR out %d: %s\n", i, hipGetErrorString(error));
 		}
      
    //  error = hipMemset3D(grids->vWFbond, -1, grids->iextent);
    //  if (error) printf("cudaERROR Memset vWFbond: %s\n", hipGetErrorString(error));
    
      		p3d.srcPtr = make_hipPitchedPtr(ux, grids->width*sizeof(double), grids->width*sizeof(double), grids->height);
      		p3d.dstPtr = grids->ux;
      		error = hipMemcpy3D(&p3d);
      		if (error) printf("cudaERROR ux: %s\n", hipGetErrorString(error));

      		p3d.srcPtr = make_hipPitchedPtr(uy, grids->width*sizeof(double), grids->width*sizeof(double), grids->height);
      		p3d.dstPtr = grids->uy;
      		error = hipMemcpy3D(&p3d);
      		if (error) printf("cudaERROR uy: %s\n", hipGetErrorString(error));

      		p3d.srcPtr = make_hipPitchedPtr(uz, grids->width*sizeof(double), grids->width*sizeof(double), grids->height);
      		p3d.dstPtr = grids->uz;
      		error = hipMemcpy3D(&p3d);
      		if (error) printf("cudaERROR uz: %s\n", hipGetErrorString(error));

      		p3d.srcPtr = make_hipPitchedPtr(rho, grids->width*sizeof(double), grids->width*sizeof(double), grids->height);
      		p3d.dstPtr = grids->rho;
      		error = hipMemcpy3D(&p3d);
      		if (error) printf("cudaERROR rho: %s\n", hipGetErrorString(error));
      
      		p3d.srcPtr = make_hipPitchedPtr(Fx, grids->width*sizeof(double), grids->width*sizeof(double), grids->height);
      		p3d.dstPtr = grids->Fx;
      		error = hipMemcpy3D(&p3d);
      		if (error) printf("cudaERROR Fx: %s\n", hipGetErrorString(error));

      		p3d.srcPtr = make_hipPitchedPtr(Fy, grids->width*sizeof(double), grids->width*sizeof(double), grids->height);
      		p3d.dstPtr = grids->Fy;
      		error = hipMemcpy3D(&p3d);
      		if (error) printf("cudaERROR Fy: %s\n", hipGetErrorString(error));

      		p3d.srcPtr = make_hipPitchedPtr(Fz, grids->width*sizeof(double), grids->width*sizeof(double), grids->height);
      		p3d.dstPtr = grids->Fz;
      		error = hipMemcpy3D(&p3d);
      		if (error) printf("cudaERROR Fz: %s\n", hipGetErrorString(error));
      
      		p3d.extent = grids->fextent;
      		p3d.srcPtr = make_hipPitchedPtr(obstacle, grids->width*sizeof(float), grids->width*sizeof(float), grids->height);
      		p3d.dstPtr = grids->obst;
      		error = hipMemcpy3D(&p3d);
      		if (error) printf("cudaERROR obstacle: %s\n", hipGetErrorString(error));
      
      		p3d.extent = grids->iextent;
      		p3d.srcPtr = make_hipPitchedPtr(vWFbond, grids->width*sizeof(int), grids->width*sizeof(int), grids->height);
      		p3d.dstPtr = grids->vWFbond;
      		error = hipMemcpy3D(&p3d);
      		if (error) printf("cudaERROR vWFbond: %s\n", hipGetErrorString(error));

      		hipMemcpy(grids->deviceStruct, grids, sizeof(fluid_GPUgrids), hipMemcpyHostToDevice);
      		cudacheck("deviceStruct");

   	 } else {

      // Copy result to host memory
     		hipMemcpy3DParms p3d = {0};
      		hipError_t error;
      		p3d.extent = grids->dextent;
      		p3d.kind = hipMemcpyDeviceToHost;

      		for (i = 0; i < 19; ++i) {
	      		p3d.srcPtr = grids->fIN[i];
	      		p3d.dstPtr = make_hipPitchedPtr(fIN[i], grids->width*sizeof(double), grids->width*sizeof(double), grids->height);
	      		error = hipMemcpy3D(&p3d);
	      		if (error) printf("cudaERROR in 0: %s\n", hipGetErrorString(error));

	     	 	p3d.srcPtr = grids->fOUT[i];
	      		p3d.dstPtr = make_hipPitchedPtr(fOUT[i], grids->width*sizeof(double), grids->width*sizeof(double), grids->height);
	      		error = hipMemcpy3D(&p3d);
	      		if (error) printf("cudaERROR out 0: %d %s\n", i, hipGetErrorString(error));
      		}

      		p3d.srcPtr = grids->ux;
      		p3d.dstPtr = make_hipPitchedPtr(ux, grids->width*sizeof(double), grids->width*sizeof(double), grids->height);
      		error = hipMemcpy3D(&p3d);
      		if (error) printf("cudaERROR ux 0: %s\n", hipGetErrorString(error));

      		p3d.srcPtr = grids->uy;
      		p3d.dstPtr = make_hipPitchedPtr(uy, grids->width*sizeof(double), grids->width*sizeof(double), grids->height);
      		error = hipMemcpy3D(&p3d);
      		if (error) printf("cudaERROR uy 0: %s\n", hipGetErrorString(error));

      		p3d.srcPtr = grids->uz;
      		p3d.dstPtr = make_hipPitchedPtr(uz, grids->width*sizeof(double), grids->width*sizeof(double), grids->height);
      		error = hipMemcpy3D(&p3d);
      		if (error) printf("cudaERROR uz 0: %s\n", hipGetErrorString(error));

      		p3d.srcPtr = grids->rho;
      		p3d.dstPtr = make_hipPitchedPtr(rho, grids->width*sizeof(double), grids->width*sizeof(double), grids->height);
      		error = hipMemcpy3D(&p3d);
      		if (error) printf("cudaERROR rho 0: %s\n", hipGetErrorString(error)); 

      		p3d.srcPtr = grids->Fx;
      		p3d.dstPtr = make_hipPitchedPtr(Fx, grids->width*sizeof(double), grids->width*sizeof(double), grids->height);
      		error = hipMemcpy3D(&p3d);
      		if (error) printf("cudaERROR Fx 0: %s\n", hipGetErrorString(error));

      		p3d.srcPtr = grids->Fy;
      		p3d.dstPtr = make_hipPitchedPtr(Fy, grids->width*sizeof(double), grids->width*sizeof(double), grids->height);
      		error = hipMemcpy3D(&p3d);
      		if (error) printf("cudaERROR Fy 0: %s\n", hipGetErrorString(error));

      		p3d.srcPtr = grids->Fz;
      		p3d.dstPtr = make_hipPitchedPtr(Fz, grids->width*sizeof(double), grids->width*sizeof(double), grids->height);
      		error = hipMemcpy3D(&p3d);
      		if (error) printf("cudaERROR Fz 0: %s\n", hipGetErrorString(error));
      
      		p3d.extent = grids->fextent;           
      		p3d.srcPtr = grids->obst;
      		p3d.dstPtr = make_hipPitchedPtr(obstacle, grids->width*sizeof(float), grids->width*sizeof(float), grids->height);
      		error = hipMemcpy3D(&p3d);
      		if (error) printf("cudaERROR obstacle 0: %s\n", hipGetErrorString(error));

    	}
}

  ///END: FLUID ALLOC AND INITIAL ///


 // Execution fluid step(s)
void fluid_invokeGPUKernel(void *model, void *g, void *g_SEM, double *randNum, gsl_rng *r, int timeSteps)
{
    	int aBank, t, i, j, k;
    	fluid_GPUgrids *grids = (fluid_GPUgrids *)g;
    	sem_GPUgrids *grids_SEM = (sem_GPUgrids *)g_SEM;
    	int SurfElem = grids_SEM->SurfElem;
    	int numReceptorsPerElem = grids_SEM->numReceptorsPerElem;
    // Z blocksPerGrid has to be == 1 so break into banks
    	int bankDepth = 16;
    	int zBanks = (grids->depth + bankDepth - 1) / bankDepth;
    	dim3 threadsPerBlock3D(32, 1, bankDepth);
    	dim3 blocksPerGrid3D((grids->width + threadsPerBlock3D.x - 1) / threadsPerBlock3D.x,
			 (grids->height + threadsPerBlock3D.y - 1) / threadsPerBlock3D.y, 1);
    
    	int threadsPerBlockSEM = 32;
    	int blocksPerGridSEM = (grids_SEM->maxElements + threadsPerBlockSEM - 1)/threadsPerBlockSEM;
    	int blocksPerGridRecept = grids_SEM->SurfElem;
    	int threadsPerBlockRecept = grids_SEM->numReceptorsPerElem;
    	hipError_t error;
   	hipMemcpy3DParms p3d = {0};
    	p3d.extent = grids_SEM->dextent;
    
    	for (t = 0; t < timeSteps; ++t) {
	  
	  	error = hipMemset3D(grids->Fx, 0, grids->dextent);//set fluid grid external force to zero.
	  	if (error) printf("cudaERROR Memset Fx: %s\n", hipGetErrorString(error));

	  	error = hipMemset3D(grids->Fy, 0, grids->dextent);
	  	if (error) printf("cudaERROR Memset Fy: %s\n", hipGetErrorString(error));

	  	error = hipMemset3D(grids->Fz, 0, grids->dextent);
	  	if (error) printf("cudaERROR Memset Fz: %s\n", hipGetErrorString(error));

	 // error = hipMemset2D(grids_SEM->rho, grids_SEM->pitch, 0, grids_SEM->maxCells * sizeof(double), grids_SEM->maxElements);
	 // if (error) printf("cudaERROR Memset Rho: %s\n", hipGetErrorString(error));
	  
          	error = hipMemset2D(grids_SEM->V_X, grids_SEM->pitch, 0, grids_SEM->maxCells * sizeof(double), grids_SEM->maxElements);//set SEM velocity to zero.
	  	if (error) printf("cudaERROR Memset Vx: %s\n", hipGetErrorString(error));

	  	error = hipMemset2D(grids_SEM->V_Y, grids_SEM->pitch, 0, grids_SEM->maxCells * sizeof(double), grids_SEM->maxElements);
	  	if (error) printf("cudaERROR Memset Vy: %s\n", hipGetErrorString(error));

	  	error = hipMemset2D(grids_SEM->V_Z, grids_SEM->pitch, 0, grids_SEM->maxCells * sizeof(double), grids_SEM->maxElements);
	  	if (error) printf("cudaERROR Memset Vz: %s\n", hipGetErrorString(error));
    
          	p3d.kind = hipMemcpyDeviceToHost;          
          	p3d.srcPtr = grids_SEM->randNum;
          	p3d.dstPtr = make_hipPitchedPtr(randNum, SurfElem*sizeof(double),SurfElem*sizeof(double), numReceptorsPerElem);
          	error = hipMemcpy3D(&p3d);
          	if (error) printf("cudaERROR randNum to host: %s\n", hipGetErrorString(error));
          	for (i = 0; i < SurfElem; i++){
              		for (j = 0; j < numReceptorsPerElem; j++){
                  		for (k = 0; k < 2; k++){
                      			if (*(randNum + k * SurfElem * numReceptorsPerElem + j * SurfElem + i) < 0)
                         			*(randNum + k * SurfElem * numReceptorsPerElem + j * SurfElem + i) = gsl_rng_uniform(r);
                  		}
              		}
          	} 
          	p3d.kind = hipMemcpyHostToDevice;
          	p3d.srcPtr = p3d.dstPtr;
          	p3d.dstPtr = grids_SEM->randNum;
          	error = hipMemcpy3D(&p3d);
          	if (error) printf("cudaERROR randNum to device: %s\n", hipGetErrorString(error));          
	   
          	sem_platelet_wall_kernel<<<blocksPerGridRecept, threadsPerBlockRecept>>>(grids->deviceStruct, grids_SEM->devImage);
         	cudacheck("platelet_wall"); 
          	for (aBank = 0; aBank < zBanks; ++aBank) {
          		fluid3d_force_distribute_kernel<<< blocksPerGrid3D, threadsPerBlock3D>>>(grids->deviceStruct, grids_SEM->devImage, aBank);
          		cudacheck("force_distribute");
          	}
	  
      		for (aBank = 0; aBank < zBanks; ++aBank) {
        		fluid3d_collision_kernel<<< blocksPerGrid3D, threadsPerBlock3D >>>(grids->deviceStruct, aBank);
        		cudacheck("collision");
      		}

      		for (aBank = 0; aBank < zBanks; ++aBank) {
        		fluid3d_stream_kernel<<< blocksPerGrid3D, threadsPerBlock3D >>>(grids->deviceStruct, aBank);
        		cudacheck("stream");
      		}
	
     /* for (aBank = 0; aBank < zBanks; ++aBank) {
        fluid3d_obst_bounce_back_kernel<<< blocksPerGrid3D, threadsPerBlock3D >>>(grids->deviceStruct, aBank);
        cudacheck("obst_bounce");
      }*/

     /* for (aBank = 0; aBank < zBanks; ++aBank) {
        fluid3d_obst_stream_kernel<<< blocksPerGrid3D, threadsPerBlock3D >>>(grids->deviceStruct, aBank);
        cudacheck("obst_stream");
      }*/
      		for (aBank = 0; aBank < zBanks; ++aBank) {
        		fluid3d_noslip_boundary_kernel<<< blocksPerGrid3D, threadsPerBlock3D >>>(grids->deviceStruct, aBank);
        		cudacheck("no_slip");
      		}
      
      		for (aBank = 0; aBank < zBanks; ++aBank) {
        		fluid3d_moving_plate_boundary_kernel<<< blocksPerGrid3D, threadsPerBlock3D >>>(grids->deviceStruct, aBank);
        		cudacheck("moving_plate");
      		}
      
   /*   for (aBank = 0; aBank < zBanks; ++aBank) {
        fluid3d_edge_corner_boundary_kernel<<< blocksPerGrid3D, threadsPerBlock3D >>>(grids->deviceStruct, aBank);
        cudacheck("edge_corner");
      }*/
      

     /* for (aBank = 0; aBank < zBanks; ++aBank) {
        fluid3d_yzplane_boundary_kernel<<< blocksPerGrid3D, threadsPerBlock3D >>>(grids->deviceStruct, aBank);
        cudacheck("yzplane");
      }*/
      
      //// DO NOT CALL fluid3d_in_out_flow_boundary_kernel before fluid3d_obst_bounce_back_kernel() 
      //// and fluid3d_obst_stream_kernel(). There are ad hoc fix in fluid3d_in_out_flow_boundary_kernel 
      //// for fluid nodes on inflow/outflow plane directly linked to solid wall nodes.
     /* for (aBank = 0; aBank < zBanks; ++aBank) {
        fluid3d_in_out_flow_boundary_kernel<<< blocksPerGrid3D, threadsPerBlock3D >>>(grids->deviceStruct, aBank);
        cudacheck("flow_boundary");
      }*/
      		for (aBank = 0; aBank < zBanks; ++aBank) {
        		fluid3d_velocity_density_kernel<<< blocksPerGrid3D, threadsPerBlock3D >>>(grids->deviceStruct, aBank);
        		cudacheck("velocity_density");
      		}
      
       		for (aBank = 0; aBank < zBanks; ++aBank) {
         		fluid3d_velocity_distribute_kernel<<< blocksPerGrid3D, threadsPerBlock3D>>>(grids->deviceStruct, grids_SEM->devImage, aBank);
         		cudacheck("velocity_distribute");
      	 	}

	  	sem_update_position_kernel<<<blocksPerGridSEM,threadsPerBlockSEM>>>(grids->deviceStruct, grids_SEM->devImage);
	  	cudacheck("update_position");

	}
    	hipMemcpy(grids_SEM, grids_SEM->devImage, sizeof(sem_GPUgrids), hipMemcpyDeviceToHost);
    	cudacheck("sem devImage to host"); 
     
}

  //Calculate force acting on elements
void sem_invokeGPUKernel_Force(void *model, void *g, int timeSteps, int* done, int* totalT , gsl_rng *r, double gama)
  //  add force if to pass back to io
{
   	int t;
   	sem_GPUgrids *grids = (sem_GPUgrids *)g;
   	double randNum;
 
   // printf("randNum = %e, gama = %e, totalBond = %d\n", randNum, gama, grids->totalBond); 
   // randNum = gsl_rng_uniform(r);
   // printf("randNum = %e, gama = %e, totalBond = %d\n", randNum, gama, grids->totalBond); 
   	if ((grids->totalBond == 0) && ((randNum = gsl_rng_uniform(r)) < gama)){  
   // if ((grids->totalBond == 0) && ((randNum) < gama)){  
      	printf("randNum = %e, gama = %e\n", randNum, gama); 
      	*done = 1;
   	} 
   	else {        
     		int threadsPerBlock = 32;
     		int threadsPerBlock_FEM = 32;
     		int threadsPerBlock_node = 32; 
     		int blocksPerGrid = (grids->maxElements + threadsPerBlock - 1)/threadsPerBlock;
     		int blocksPerGrid_FEM = (grids->SurfElem + threadsPerBlock_FEM - 1)/threadsPerBlock_FEM;
     		int blocksPerGrid_node =(grids->newnodeNum + threadsPerBlock_node - 1)/threadsPerBlock_node;
     		hipError_t error;
     		(*totalT)++;
 
     		for (t = 0; t < timeSteps; ++t) {
          		grids->S_all = 0;
          		grids->V = 0;
         	 	grids->totalBond = 0;
         // printf("S = %e\n", grids->S_all);
          		error = hipMemset(grids->S, 0, grids->SurfElem * sizeof(double));//set SEM velocity to zero.
	  		if (error) printf("cudaERROR Memset S: %s\n", hipGetErrorString(error));
          
          		error = hipMemset2D(grids->F_X, grids->pitch, 0, grids->maxCells * sizeof(double), grids->maxElements);//set SEM velocity to zero.
	  		if (error) printf("cudaERROR Memset F_x: %s\n", hipGetErrorString(error));

	  		error = hipMemset2D(grids->F_Y, grids->pitch, 0, grids->maxCells * sizeof(double), grids->maxElements);
	  		if (error) printf("cudaERROR Memset F_y: %s\n", hipGetErrorString(error));

	  		error = hipMemset2D(grids->F_Z, grids->pitch, 0, grids->maxCells * sizeof(double), grids->maxElements);
	  		if (error) printf("cudaERROR Memset F_z: %s\n", hipGetErrorString(error));
	  
          		error = hipMemset2D(grids->n, grids->pitch, 0, 3 * sizeof(double), grids->newnodeNum);
	  		if (error) printf("cudaERROR Memset n: %s\n", hipGetErrorString(error));
          
          		error = hipMemset2D(grids->nelem, grids->pitch, 0, 3 * sizeof(double), grids->SurfElem);
	  		if (error) printf("cudaERROR Memset nelem: %s\n", hipGetErrorString(error));
          
          		error = hipMemset2D(grids->q, grids->pitch, 0, 3 * sizeof(double), grids->newnodeNum);
	  		if (error) printf("cudaERROR Memset q: %s\n", hipGetErrorString(error));
          
          		error = hipMemset2D(grids->A, grids->pitch, 0, 9 * sizeof(double), grids->newnodeNum);
          		if (error) printf("cudaERROR Memset A: %s\n", hipGetErrorString(error));
          
          		error = hipMemset2D(grids->tau, grids->pitch, 0, 9 * sizeof(double), grids->newnodeNum);
          		if (error) printf("cudaERROR Memset tau: %s\n", hipGetErrorString(error));

          		error = hipMemset2D(grids->Kapa, grids->pitch, 0, 9 * sizeof(double), grids->newnodeNum);
          		if (error) printf("cudaERROR Memset Kapa: %s\n", hipGetErrorString(error));

	  		error = hipMemset2D(grids->K, grids->pitch, 0, 27 * sizeof(double), grids->newnodeNum);
	  		if (error) printf("cudaERROR Memset K: %s\n", hipGetErrorString(error));
	  
          		error = hipMemset2D(grids->Laplace_km, grids->pitch, 0, sizeof(double), grids->maxElements);
	  		if (error) printf("cudaERROR Memset Laplace_km: %s\n", hipGetErrorString(error));

          		hipMemcpy(grids->devImage, grids, sizeof(sem_GPUgrids), hipMemcpyHostToDevice);
          		cudacheck("devImage"); 
        
          		sem_Force_kernel<<< blocksPerGrid, threadsPerBlock >>>(grids->devImage);
          		cudacheck("sem_Force");
          		sem_calculate_A_kernel<<<blocksPerGrid_FEM, threadsPerBlock_FEM>>>(grids->devImage);
          		cudacheck("calculate_A");        
          		sem_surface_tension_kernel<<< blocksPerGrid_node, threadsPerBlock_node >>>(grids->devImage);
          		cudacheck("sem_surface_tension");
          		sem_calculate_Kapa_kernel<<<blocksPerGrid_FEM, threadsPerBlock_FEM>>>(grids->devImage);        
          		sem_calculate_m_kernel<<< blocksPerGrid_node, threadsPerBlock_node >>>(grids->devImage);
        //sem_calculate_K_kernel<<<blocksPerGrid_FEM, threadsPerBlock_FEM>>>(grids->devImage);        
        //sem_calculate_q_kernel<<< blocksPerGrid_node, threadsPerBlock_node >>>(grids->devImage);
        //sem_bending_force_kernel<<<blocksPerGrid_FEM, threadsPerBlock_FEM>>>(grids->devImage);               
        //cudacheck("bending_force");
          		sem_ForceAreaVol_kernel<<<blocksPerGrid_FEM, threadsPerBlock_FEM>>>(grids->devImage);
          		sem_Laplace_Kapa_kernel<<< blocksPerGrid, threadsPerBlock >>>(grids->devImage);
          		cudacheck("sem_Laplace_Kapa");         
          
          //hipMemcpy(grids, grids->devImage, sizeof(sem_GPUgrids), hipMemcpyDeviceToHost);
          //cudacheck("devImage to host"); 
		}  
     	}

} //// END: sem_invokeGPUKernel_Force() 

// Release
void fluid_releaseGPUKernel(void *model, void *g)
{
	fluid_GPUgrids *grids = (fluid_GPUgrids *)g;
       int i;
       for (i = 0; i < 19; ++i) {
         hipFree(&(grids->fIN[i]));
         hipFree(&(grids->fOUT[i]));
       }
       hipFree(&(grids->ux));
       hipFree(&(grids->uy));
       hipFree(&(grids->uz));
       hipFree(&(grids->obst));
       hipFree(&(grids->rho));
       hipFree(&(grids->Fx));
       hipFree(&(grids->Fy));
       hipFree(&(grids->Fz));
       hipFree(&(grids->vWFbond));
       hipFree(grids->deviceStruct);
       free(grids);
       hipDeviceReset();
}

///////////End of GPU Fluid ///////////////////////////////////////////////////////////////////


////////// Begin of the GPU for Fiber Network //////////////////////////////////////////////////
////////// Fibre alloc and initial///////
// alloc 
void *fiber_allocGPUKernel(void *model, int maxNodes, int maxLinks,
                           int max_N_conn_at_Node,double dt, double *hostParameters)
{
	tmp_fiber_GPUgrids *fgrids=(tmp_fiber_GPUgrids *)malloc(sizeof(tmp_fiber_GPUgrids));
	fgrids->maxNodes = maxNodes;
	fgrids->maxLinks = maxLinks;
	fgrids->dt = dt;
	fgrids->max_N_conn_at_Node = max_N_conn_at_Node;

        //  allocate memory for fiber nodes //////
        hipMalloc((void**)&(fgrids->X), maxNodes * sizeof(double));	
        hipMalloc((void**)&(fgrids->Y), maxNodes * sizeof(double));	
        hipMalloc((void**)&(fgrids->Z), maxNodes * sizeof(double));	

        hipMalloc((void**)&(fgrids->X0), maxNodes * sizeof(double));	
        hipMalloc((void**)&(fgrids->Y0), maxNodes * sizeof(double));	
        hipMalloc((void**)&(fgrids->Z0), maxNodes * sizeof(double));	

        hipMalloc((void**)&(fgrids->V_X), maxNodes * sizeof(double));	
        hipMalloc((void**)&(fgrids->V_Y), maxNodes * sizeof(double));	
        hipMalloc((void**)&(fgrids->V_Z), maxNodes * sizeof(double));	

        hipMalloc((void**)&(fgrids->F_X), maxNodes * sizeof(double));	
        hipMalloc((void**)&(fgrids->F_Y), maxNodes * sizeof(double));	
        hipMalloc((void**)&(fgrids->F_Z), maxNodes * sizeof(double));	

        hipMalloc((void**)&(fgrids->NodeType), maxNodes * sizeof(int));	
        hipMalloc((void**)&(fgrids->N_Conn_at_Node), maxNodes * sizeof(int));	


        //  allocate memory for fiber link //////
        hipMallocPitch((void**)&(fgrids->Link_at_Node), &(fgrids->pitchLink_at_Node), max_N_conn_at_Node*sizeof(int), maxNodes); 
        hipMallocPitch((void**)&(fgrids->lAdjVer), &(fgrids->pitchlAdjVer), 2*sizeof(int),  maxLinks); 

        hipMalloc((void**)&(fgrids->linkLengths), maxLinks * sizeof(double));	
        hipMalloc((void**)&(fgrids->linkLengths0), maxLinks * sizeof(double));	
        hipMalloc((void**)&(fgrids->linkThick), maxLinks * sizeof(double));	

        return fgrids;
}


	


//// INITIAL ///////////////////////////////////////////////////////////////////
  
void fiber_init_GPUKernel(void *model, void *fg, void *NodeType, void *N_Conn_at_Node, 
                           void *Link_at_Node, void *lAdjVer, void *linkLengths,void *linkLengths0, 
                           void *linkThick, void *X,void *V_X,void *X0,  void *F_X, 
                           void *Y,void *Y0, void *V_Y, void *F_Y, void *Z,void *Z0, void *V_Z, void *F_Z )
{
     	tmp_fiber_GPUgrids *fgrids = (tmp_fiber_GPUgrids *)fg;
  
    // memory copy for node
   	hipMemcpy(fgrids->N_Conn_at_Node, N_Conn_at_Node, fgrids->maxNodes * sizeof(int), hipMemcpyHostToDevice);
    	cudacheck("N_Conn_at_Node");

   	hipMemcpy(fgrids->NodeType, NodeType, fgrids->maxNodes * sizeof(int), hipMemcpyHostToDevice);
    	cudacheck("NodeType");
  	hipMemcpy(fgrids->X, X, fgrids->maxNodes * sizeof(double), hipMemcpyHostToDevice);
    	cudacheck("X");
  	hipMemcpy(fgrids->Y, Y, fgrids->maxNodes * sizeof(double), hipMemcpyHostToDevice);
    	cudacheck("Y");
   	hipMemcpy(fgrids->Z, Z, fgrids->maxNodes * sizeof(double), hipMemcpyHostToDevice);
    	cudacheck("Z");
   
  	hipMemcpy(fgrids->X0, X0, fgrids->maxNodes * sizeof(double), hipMemcpyHostToDevice);
    	cudacheck("X0");
   	hipMemcpy(fgrids->Y0, Y0, fgrids->maxNodes * sizeof(double), hipMemcpyHostToDevice);
    	cudacheck("Y0");
   	hipMemcpy(fgrids->Z0, Z0, fgrids->maxNodes * sizeof(double), hipMemcpyHostToDevice);
    	cudacheck("Z0");

   	hipMemcpy(fgrids->V_X, V_X, fgrids->maxNodes * sizeof(double), hipMemcpyHostToDevice);
    	cudacheck("V_X");
   	hipMemcpy(fgrids->V_Y, V_Y, fgrids->maxNodes * sizeof(double), hipMemcpyHostToDevice);
    	cudacheck("V_Y");
   	hipMemcpy(fgrids->V_Z, V_Z, fgrids->maxNodes * sizeof(double), hipMemcpyHostToDevice);
    	cudacheck("V_Z");
  
   	hipMemcpy(fgrids->F_X, F_X, fgrids->maxNodes * sizeof(double), hipMemcpyHostToDevice);
    	cudacheck("F_X");
   	hipMemcpy(fgrids->F_Y, F_Y, fgrids->maxNodes * sizeof(double), hipMemcpyHostToDevice);
    	cudacheck("F_Y");
   	hipMemcpy(fgrids->F_Z, F_Z, fgrids->maxNodes * sizeof(double), hipMemcpyHostToDevice);
    	cudacheck("F_Z");
  // memory copy for the link 

   	hipMemcpy2D(fgrids->Link_at_Node, fgrids->pitchLink_at_Node, Link_at_Node, fgrids->max_N_conn_at_Node * sizeof(int), fgrids->max_N_conn_at_Node * sizeof(int), fgrids->maxNodes, hipMemcpyHostToDevice);
   	cudacheck("Link_at_Node");
   	hipMemcpy2D(fgrids->lAdjVer, fgrids->pitchlAdjVer, lAdjVer, 2 * sizeof(int), 2 * sizeof(int), fgrids->maxLinks, hipMemcpyHostToDevice);
   	cudacheck("lAdjVer");

   	hipMemcpy(fgrids->linkLengths, linkLengths, fgrids->maxLinks * sizeof(double), hipMemcpyHostToDevice);
    	cudacheck("linkLengths");
   	hipMemcpy(fgrids->linkLengths0, linkLengths0, fgrids->maxLinks * sizeof(double), hipMemcpyHostToDevice);
    	cudacheck("linkLengths0");
   	hipMemcpy(fgrids->linkThick, linkThick, fgrids->maxLinks * sizeof(double), hipMemcpyHostToDevice);
    	cudacheck("linkThick");
  
  
}


// COMPUTE THE FORCE ///////////

void fiber_copy_GPUKernel(void *model, void *fg,  void *X,void *V_X,void *X0,  void *F_X, 
                           void *Y,void *Y0, void *V_Y, void *F_Y, void *Z,void *Z0, void *V_Z, void *F_Z )
{
     	tmp_fiber_GPUgrids *fgrids = (tmp_fiber_GPUgrids *)fg;

	hipMemcpy(X,fgrids->X, fgrids->maxNodes * sizeof(double), hipMemcpyDeviceToHost);
  	hipMemcpy(Y,fgrids->Y, fgrids->maxNodes * sizeof(double), hipMemcpyDeviceToHost);
   	hipMemcpy(Z,fgrids->Z, fgrids->maxNodes * sizeof(double), hipMemcpyDeviceToHost);
    	
	hipMemcpy(V_X,fgrids->V_X, fgrids->maxNodes * sizeof(double), hipMemcpyDeviceToHost);
  	hipMemcpy(V_Y,fgrids->V_Y, fgrids->maxNodes * sizeof(double), hipMemcpyDeviceToHost);
   	hipMemcpy(V_Z,fgrids->V_Z, fgrids->maxNodes * sizeof(double), hipMemcpyDeviceToHost);

	hipMemcpy(F_X,fgrids->F_X, fgrids->maxNodes * sizeof(double), hipMemcpyDeviceToHost);
  	hipMemcpy(F_Y,fgrids->F_Y, fgrids->maxNodes * sizeof(double), hipMemcpyDeviceToHost);
   	hipMemcpy(F_Z,fgrids->F_Z, fgrids->maxNodes * sizeof(double), hipMemcpyDeviceToHost);
	




}
// memory copy from device to host 














// memory release 

void fiber_release_GPUKernel(void *model, void *fg)
{	
	tmp_fiber_GPUgrids *fgrids = (tmp_fiber_GPUgrids *)fg;

    	
    	hipFree(fgrids->NodeType);
    	hipFree(fgrids->N_Conn_at_Node);
    	hipFree(fgrids->Link_at_Node);
    	hipFree(fgrids->lAdjVer);
    	hipFree(fgrids->linkLengths);
    	hipFree(fgrids->linkLengths0);
    	hipFree(fgrids->linkThick);
    
    	hipFree(fgrids->X);
    	hipFree(fgrids->V_X);
    	hipFree(fgrids->F_X);
   
  	hipFree(fgrids->Y);
    	hipFree(fgrids->V_Y);
    	hipFree(fgrids->F_Y);
   
	hipFree(fgrids->Z);
   	hipFree(fgrids->V_Z);
    	hipFree(fgrids->F_Z);

    //hipFree(grids->cellCenterZ);
    free(fgrids);
    hipDeviceReset();
}



//////////////////////////END of GPU for the Fiber Network /////////////////////////////////////




/////////////////////////BEGIN OF GPU for the Platelets ///////////////////////////////////////

  ///BEGIN: SEM ALLOC AND INITIAL ///
  // Allocation
void *sem_allocGPUKernel(void *model, int maxCells, int maxElements, int SurfElem, 
                           int newnode, int numReceptorsPerElem, 
                           float dt, double S0_all, float *hostParameters)
{
   	sem_GPUgrids *grids = (sem_GPUgrids *)malloc(sizeof(sem_GPUgrids));

   // Save parameters
   	grids->maxCells = maxCells;
   	grids->maxElements = maxElements;
   	grids->dt = dt;
   	grids->SurfElem = SurfElem;
	grids->newnodeNum = newnode;
   	grids->numReceptorsPerElem = numReceptorsPerElem;
   	grids->S0_all = S0_all;
   	grids->iextent = make_hipExtent(grids->SurfElem*sizeof(int), grids->numReceptorsPerElem, 3);
   	grids->dextent = make_hipExtent(grids->SurfElem*sizeof(double), grids->numReceptorsPerElem, 2);
   
  // grids->ReversalPeriod = ReversalPeriod;
   // Allocate device memory
   	hipMalloc((void**)&(grids->devImage),sizeof(sem_GPUgrids));
   	size_t pitch_test;
   // cells and elements
   	hipMalloc((void**)&(grids->numOfElements), maxCells * sizeof(int));
   	hipMalloc((void**)&(grids->node_nbrElemNum), newnode * sizeof(int));
   	hipMalloc((void**)&(grids->S0), SurfElem * sizeof(double));
   	hipMalloc((void**)&(grids->S), SurfElem * sizeof(double));
   
   	hipMallocPitch((void**)&(grids->elementType), &(grids->pitch), maxCells * sizeof(int), maxElements);
   	pitch_test = grids->pitch;
   	hipMallocPitch((void**)&(grids->triElem), &(grids->pitch), 6 * sizeof(int), SurfElem); 
   	pitchcheck(pitch_test, grids->pitch, 1);
   	hipMallocPitch((void**)&(grids->receptor_r1), &(grids->pitch), numReceptorsPerElem * sizeof(float), SurfElem); 
   	pitchcheck(pitch_test, grids->pitch, 2);
   	hipMallocPitch((void**)&(grids->receptor_r2), &(grids->pitch), numReceptorsPerElem * sizeof(float), SurfElem);
   	pitchcheck(pitch_test, grids->pitch, 3);
   //hipMallocPitch((void**)&(grids->rho), &(grids->pitch), maxCells * sizeof(double), maxElements);
   //pitchcheck(pitch_test, grids->pitch);
   	hipMallocPitch((void**)&(grids->X_Ref), &(grids->pitch), maxCells * sizeof(double), maxElements);
   	pitchcheck(pitch_test, grids->pitch, 4);
   	hipMallocPitch((void**)&(grids->X), &(grids->pitch), maxCells * sizeof(double), maxElements);
   	pitchcheck(pitch_test, grids->pitch, 5);
   	hipMallocPitch((void**)&(grids->V_X), &(grids->pitch), maxCells * sizeof(double), maxElements);
   	pitchcheck(pitch_test, grids->pitch, 6);
   	hipMallocPitch((void**)&(grids->F_X), &(grids->pitch), maxCells * sizeof(double), maxElements);
   	pitchcheck(pitch_test, grids->pitch, 7);
   	hipMallocPitch((void**)&(grids->Y_Ref), &(grids->pitch), maxCells * sizeof(double), maxElements);
   	pitchcheck(pitch_test, grids->pitch, 8);
   	hipMallocPitch((void**)&(grids->Y), &(grids->pitch), maxCells * sizeof(double), maxElements);
   	pitchcheck(pitch_test, grids->pitch, 9);
   	hipMallocPitch((void**)&(grids->V_Y), &(grids->pitch), maxCells * sizeof(double), maxElements);
   	pitchcheck(pitch_test, grids->pitch, 10);
   	hipMallocPitch((void**)&(grids->RY), &(grids->pitch), maxCells * sizeof(double), maxElements);
   	pitchcheck(pitch_test, grids->pitch, 11);
   	hipMallocPitch((void**)&(grids->F_Y), &(grids->pitch), maxCells * sizeof(double), maxElements);
   	pitchcheck(pitch_test, grids->pitch, 12);
   	hipMallocPitch((void**)&(grids->Z_Ref), &(grids->pitch), maxCells * sizeof(double), maxElements);
   	pitchcheck(pitch_test, grids->pitch, 13);
   	hipMallocPitch((void**)&(grids->Z), &(grids->pitch), maxCells * sizeof(double), maxElements);
   	pitchcheck(pitch_test, grids->pitch, 14);
   	hipMallocPitch((void**)&(grids->V_Z), &(grids->pitch), maxCells * sizeof(double), maxElements);
   	pitchcheck(pitch_test, grids->pitch, 15);
   	hipMallocPitch((void**)&(grids->F_Z), &(grids->pitch), maxCells * sizeof(double), maxElements);
   	pitchcheck(pitch_test, grids->pitch, 16);
   	hipMallocPitch((void**)&(grids->node_share_Elem), &(grids->pitch), 10 * sizeof(int), newnode);
   	pitchcheck(pitch_test, grids->pitch, 17);
   	hipMallocPitch((void**)&(grids->N), &(grids->pitch), 3 * sizeof(double), newnode);
   	pitchcheck(pitch_test, grids->pitch, 18);
   	hipMallocPitch((void**)&(grids->n), &(grids->pitch), 3 * sizeof(double), newnode);
   	pitchcheck(pitch_test, grids->pitch, 19);
   	hipMallocPitch((void**)&(grids->q), &(grids->pitch), 3 * sizeof(double), newnode);
   	pitchcheck(pitch_test, grids->pitch, 20);
   	hipMallocPitch((void**)&(grids->A), &(grids->pitch), 9 * sizeof(double), newnode);
   	pitchcheck(pitch_test, grids->pitch, 21);
   	hipMallocPitch((void**)&(grids->tau), &(grids->pitch), 9 * sizeof(double), newnode);
   	pitchcheck(pitch_test, grids->pitch,22);
   	hipMallocPitch((void**)&(grids->Kapa), &(grids->pitch), 9 * sizeof(double), newnode);
   	pitchcheck(pitch_test, grids->pitch, 23);
   	hipMallocPitch((void**)&(grids->km), &(grids->pitch),sizeof(double), newnode);
   	pitchcheck(pitch_test, grids->pitch, 24);
   	hipMallocPitch((void**)&(grids->K), &(grids->pitch), 27 * sizeof(double), newnode);
   	pitchcheck(pitch_test, grids->pitch, 25); 
   	hipMallocPitch((void**)&(grids->nelem), &(grids->pitch), 3 * sizeof(double), SurfElem);
   	pitchcheck(pitch_test, grids->pitch, 26); 
   	hipMallocPitch((void**)&(grids->Laplace_km), &(grids->pitch),sizeof(double), maxElements);
   	pitchcheck(pitch_test, grids->pitch, 27);
   	hipMallocPitch((void**)&(grids->node_nbrNodes), &(grids->pitch), 10 * sizeof(int), maxElements);
   	pitchcheck(pitch_test, grids->pitch, 28);
   	hipMallocPitch((void**)&(grids->bondLengths), &(grids->bondpitch), maxElements * sizeof(double), maxElements);
   	pitchcheck(pitch_test, grids->pitch, 29);

   	hipMalloc3D(&(grids->receptBond), grids->iextent);
   	hipMalloc3D(&(grids->randNum), grids->dextent);
   	cudacheck("sem_alloc");
   

  // Reversal Clock Values of cells
//   hipMalloc((void**)&(grids->ClockValue), maxCells * sizeof(int));
 //  hipMalloc((void**)&(grids->SlimeDir), maxCells * sizeof(int));

   // cell centers
  // hipMalloc((void**)&(grids->cellCenterX), maxCells * sizeof(float));
  // hipMalloc((void**)&(grids->cellCenterY), maxCells * sizeof(float));
  // hipMalloc((void**)&(grids->cellCenterZ), maxCells * sizeof(float));

   // copy parameters
   	hipMemcpyToSymbol(HIP_SYMBOL(model_Parameters), hostParameters, 100 * sizeof(float), 0, hipMemcpyHostToDevice);
 
   //  Allocate Memory for RNG states
   /* Allocate space for prng states on device */
   	hipMalloc((void **)&(grids->devState), SurfElem * numReceptorsPerElem *sizeof(hiprandState));

   	return grids;
}


// Initialization
void sem_initGPUKernel(void *model, void *g, int numOfCells, int *numOfElements, int SurfElem, int numReceptorsPerElem,
                          void *hostX_Ref, void *hostY_Ref, void *hostZ_Ref,
                          void *hostX, void *hostY, void *hostRY, void *hostZ, void *hostVX, void *hostVY, void *hostVZ,
                          void *hostFX, void *hostFY, void *hostFZ, void *hostType, void *hostBonds, 
                          void *triElem, void *receptor_r1, void *receptor_r2, void *node_share_Elem, void *N,
                          void * node_nbrElemNum, void * node_nbr_nodes, void *S0, double V0, void *receptBond, void *randNum)//transfer function  // add hostV_X...hostF_X
{


    	sem_GPUgrids *grids = (sem_GPUgrids *)g;

   // Begin RNG Stuff
   /* Setup prng states */
   // printf("SurfElem = %d, numReceptors = %d\n", SurfElem, numReceptorsPerElem);
     // setup_RNG_kernel<<<SurfElem, numReceptorsPerElem>>>(grids->devState);
   // RNG finished
   // cudacheck("setup_RNG_kernel");
    	grids->numOfCells = numOfCells;
    	grids->S_all = 0;
    	grids->V0 = V0;
    	grids->V = 0;
    	grids->totalBond = 1;
   // grids->SurfElem = SurfElem;
   // grids->numReceptorsPerElem = numReceptorsPerElem;

    // Copy host memory to device memory
    	hipMemcpy(grids->numOfElements, numOfElements, grids->maxCells * sizeof(int), hipMemcpyHostToDevice);
    	cudacheck("numOfElements"); 
    	hipMemcpy(grids->node_nbrElemNum, node_nbrElemNum, grids->newnodeNum * sizeof(int), hipMemcpyHostToDevice);
    	cudacheck("Neighbor Elements Number"); 
    	hipMemcpy(grids->S0, S0, grids->SurfElem * sizeof(double), hipMemcpyHostToDevice);
    	cudacheck("Area"); 
   // hipMemcpy(grids->ClockValue, ClockValue, grids->maxCells * sizeof(int), hipMemcpyHostToDevice);
   // hipMemcpy(grids->SlimeDir, SlimeDir, grids->maxCells * sizeof(int), hipMemcpyHostToDevice);
    	hipMemcpy(grids->devImage, grids, sizeof(sem_GPUgrids), hipMemcpyHostToDevice);
   	cudacheck("devImage"); 

    	hipMemcpy2D(grids->elementType, grids->pitch, hostType, grids->maxCells * sizeof(int), grids->maxCells * sizeof(int), grids->maxElements, hipMemcpyHostToDevice);
   	cudacheck("elementType"); 
    	hipMemcpy2D(grids->bondLengths, grids->bondpitch, hostBonds, grids->maxElements * sizeof(double), grids->maxElements * sizeof(double), grids->maxElements, hipMemcpyHostToDevice);
   	cudacheck("bondLengths"); 
    	hipMemcpy2D(grids->triElem, grids->pitch, triElem, 6 * sizeof(int), 6 * sizeof(int), grids->SurfElem, hipMemcpyHostToDevice);
   	cudacheck("triElem"); 
    	hipMemcpy2D(grids->receptor_r1, grids->pitch, receptor_r1, numReceptorsPerElem * sizeof(float), numReceptorsPerElem * sizeof(float), SurfElem, hipMemcpyHostToDevice);
   	cudacheck("receptor_r1"); 
    	hipMemcpy2D(grids->receptor_r2, grids->pitch, receptor_r2, numReceptorsPerElem * sizeof(float), numReceptorsPerElem * sizeof(float), SurfElem, hipMemcpyHostToDevice);
   	cudacheck("receptor_r2"); 
    	hipMemcpy2D(grids->X_Ref, grids->pitch, hostX_Ref, grids->maxCells * sizeof(double), grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyHostToDevice);
   	cudacheck("X_Ref"); 
    	hipMemcpy2D(grids->Y_Ref, grids->pitch, hostY_Ref, grids->maxCells * sizeof(double), grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyHostToDevice);
   	cudacheck("Y_Ref"); 
    	hipMemcpy2D(grids->Z_Ref, grids->pitch, hostZ_Ref, grids->maxCells * sizeof(double), grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyHostToDevice);
   	cudacheck("Z_Ref"); 
    	hipMemcpy2D(grids->X, grids->pitch, hostX, grids->maxCells * sizeof(double), grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyHostToDevice);
   	cudacheck("X"); 
    	hipMemcpy2D(grids->Y, grids->pitch, hostY, grids->maxCells * sizeof(double), grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyHostToDevice);
   	cudacheck("Y"); 
    	hipMemcpy2D(grids->RY, grids->pitch, hostY, grids->maxCells * sizeof(double), grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyHostToDevice);
   	cudacheck("RY"); 
    	hipMemcpy2D(grids->Z, grids->pitch, hostZ, grids->maxCells * sizeof(double), grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyHostToDevice);
   	cudacheck("Z"); 
   // added Memcpy for force and velocity
    	hipMemcpy2D(grids->F_X, grids->pitch, hostFX, grids->maxCells * sizeof(double), grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyHostToDevice);
   	cudacheck("FX"); 
    	hipMemcpy2D(grids->F_Y, grids->pitch, hostFY, grids->maxCells * sizeof(double), grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyHostToDevice);
   	cudacheck("FY"); 
    	hipMemcpy2D(grids->F_Z, grids->pitch, hostFZ, grids->maxCells * sizeof(double), grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyHostToDevice);
   	cudacheck("FZ"); 
    	hipMemcpy2D(grids->V_X, grids->pitch, hostVX, grids->maxCells * sizeof(double), grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyHostToDevice);
    	cudacheck("VX"); 
    	hipMemcpy2D(grids->V_Y, grids->pitch, hostVY, grids->maxCells * sizeof(double), grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyHostToDevice);
    	cudacheck("VY"); 
    	hipMemcpy2D(grids->V_Z, grids->pitch, hostVZ, grids->maxCells * sizeof(double), grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyHostToDevice);
    	cudacheck("VZ");
    	hipMemcpy2D(grids->node_share_Elem, grids->pitch, node_share_Elem, 10 * sizeof(int), 10 * sizeof(int), grids->newnodeNum, hipMemcpyHostToDevice);
    	cudacheck("node_share_Elem");
    	hipMemcpy2D(grids->node_nbrNodes, grids->pitch, node_nbr_nodes, 10 * sizeof(int), 10 * sizeof(int), grids->maxElements, hipMemcpyHostToDevice);
    	cudacheck("node_nbr_nodes");
    	hipMemcpy2D(grids->N, grids->pitch, N, 3 * sizeof(double), 3 * sizeof(double), grids->newnodeNum, hipMemcpyHostToDevice);
    	cudacheck("N");
   // hipMemcpy2D(grids->PreV_X, grids->pitch, hostPreVX, grids->maxCells * sizeof(double), grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyHostToDevice);
   // hipMemcpy2D(grids->PreV_Y, grids->pitch, hostPreVY, grids->maxCells * sizeof(float), grids->maxCells * sizeof(float), grids->maxElements, hipMemcpyHostToDevice);
   // hipMemcpy2D(grids->PreV_Z, grids->pitch, hostPreVZ, grids->maxCells * sizeof(float), grids->maxCells * sizeof(float), grids->maxElements, hipMemcpyHostToDevice);
   //add cudaMec.. for VX, VY, VZ, FX, FY, FZ
  // hipMemset2D((void**)&(grids->receptBond), grids->pitch, 0,  numReceptorsPerElem * sizeof(float), SurfElem); 
   // hipError_t error = hipMemset3D(grids->receptBond, -1, grids->iextent);
   // if (error) printf("cudaERROR Memset receptBond: %s\n", hipGetErrorString(error));
    
    	hipError_t error = hipMemset2D(grids->km, grids->pitch, 0, grids->maxCells * sizeof(double), grids->newnodeNum);
    	if (error) printf("cudaERROR Memset km: %s\n", hipGetErrorString(error));
    
    	hipMemcpy3DParms p3d = {0};
    	p3d.extent = grids->iextent;
    	p3d.kind = hipMemcpyHostToDevice;
      
    	p3d.srcPtr = make_hipPitchedPtr(receptBond, grids->SurfElem*sizeof(int), grids->SurfElem*sizeof(int), grids->numReceptorsPerElem);
    	p3d.dstPtr = grids->receptBond;
    	error = hipMemcpy3D(&p3d);
    	if (error) printf("cudaERROR receptBond: %s\n", hipGetErrorString(error));
    
    	p3d.extent = grids->dextent;
    	p3d.kind = hipMemcpyHostToDevice;
      
    	p3d.srcPtr = make_hipPitchedPtr(randNum, grids->SurfElem*sizeof(double), grids->SurfElem*sizeof(double), grids->numReceptorsPerElem);
    	p3d.dstPtr = grids->randNum;
    	error = hipMemcpy3D(&p3d);
    	if (error) printf("cudaERROR randNum: %s\n", hipGetErrorString(error));
      
    	cudacheck("sem_init"); 
}
///END SEM ALLOC AND INITIAL ///

 

void sem_copyGPUKernel(void *model, void *g, void *hostX, void *hostY, void *hostRY, void *hostZ, 
                          void *hostVX, void *hostVY, void *hostVZ, 
                          void *hostFX, void *hostFY, void *hostFZ, int timeSteps)
{
    // Copy result to host memory
    	sem_GPUgrids *grids = (sem_GPUgrids *)g;
    	hipMemcpy2D(hostX, grids->maxCells * sizeof(double), grids->X, grids->pitch, grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyDeviceToHost);
    	hipMemcpy2D(hostY, grids->maxCells * sizeof(double), grids->Y, grids->pitch, grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyDeviceToHost);
    	hipMemcpy2D(hostRY, grids->maxCells * sizeof(double), grids->RY, grids->pitch, grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyDeviceToHost);
    	hipMemcpy2D(hostZ, grids->maxCells * sizeof(double), grids->Z, grids->pitch, grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyDeviceToHost);

    	hipMemcpy2D(hostVX, grids->maxCells * sizeof(double), grids->V_X, grids->pitch, grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyDeviceToHost);
    	hipMemcpy2D(hostVY, grids->maxCells * sizeof(double), grids->V_Y, grids->pitch, grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyDeviceToHost);
    	hipMemcpy2D(hostVZ, grids->maxCells * sizeof(double), grids->V_Z, grids->pitch, grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyDeviceToHost);

    	hipMemcpy2D(hostFX, grids->maxCells * sizeof(double), grids->F_X, grids->pitch, grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyDeviceToHost);
    	hipMemcpy2D(hostFY, grids->maxCells * sizeof(double), grids->F_Y, grids->pitch, grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyDeviceToHost);
    	hipMemcpy2D(hostFZ, grids->maxCells * sizeof(double), grids->F_Z, grids->pitch, grids->maxCells * sizeof(double), grids->maxElements, hipMemcpyDeviceToHost);

//    hipMemcpy(ClockValue, grids->ClockValue, grids->maxCells * sizeof(int), hipMemcpyDeviceToHost);
//    hipMemcpy(SlimeDir  , grids->SlimeDir  , grids->maxCells * sizeof(int), hipMemcpyDeviceToHost);
}


   // Release
   
void sem_releaseGPUKernel(void *model, void *g)
{
    	sem_GPUgrids *grids = (sem_GPUgrids *)g;
    	hipFree(grids->numOfElements);
    	hipFree(grids->elementType);
    	hipFree(grids->bondLengths);
    	hipFree(grids->triElem);
    	hipFree(grids->receptor_r1);
    	hipFree(grids->receptor_r2);
    	hipFree(&(grids->receptBond));
    
    	hipFree(grids->X);
    	hipFree(grids->V_X);
   // hipFree(grids->PreV_X);
    	hipFree(grids->F_X);
    	hipFree(grids->Y);
    	hipFree(grids->V_Y);
    	hipFree(grids->RY);
    	hipFree(grids->F_Y);
    	hipFree(grids->Z);
    	hipFree(grids->V_Z);
   // hipFree(grids->PreV_Z);
    	hipFree(grids->F_Z);
    	hipFree(grids->devImage);
    	hipFree(grids->node_share_Elem);
    	hipFree(grids->N);
    //hipFree(grids->cellCenterZ);
    	free(grids);
    	hipDeviceReset();
}

} /// END::: extern "C" {

//////////////////////////////////End of GPU for the Platelets///////////////////////////////////
